#include "hip/hip_runtime.h"
/* 
 * pairwise_loss_layer.cu 
 * 
 *  Created on: Jan 3, 2017 
 *      Author: Limbo 
 */  
  
#include <algorithm>
#include <cfloat>  
#include <vector>  
  #include "caffe/util/io.hpp"
#include "caffe/layers/pairwise_loss_layer.hpp"  
#include "caffe/util/math_functions.hpp"  
  
namespace caffe {  
  
template <typename Dtype>  
void PairwiseLossLayer<Dtype>::Forward_gpu(  
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {  
  const int count = bottom[0]->count();  
  caffe_gpu_sub(  
      count,  
      bottom[0]->gpu_data(),  
      bottom[1]->gpu_data(),  
      diff_ap_.mutable_gpu_data()); 
  caffe_gpu_sub(  
      count,  
      bottom[2]->gpu_data(),  
      bottom[3]->gpu_data(),  
      diff_wn_.mutable_gpu_data()); 
  caffe_gpu_sub(  
      count,  
      bottom[1]->gpu_data(),  
      bottom[2]->gpu_data(),  
      diff_pn_.mutable_gpu_data());  
  
  caffe_gpu_powx(  
      count,  
      diff_ap_.mutable_gpu_data(),  
      Dtype(2),  
      diff_sq_ap_.mutable_gpu_data());  
  caffe_gpu_gemv(  
      CblasNoTrans,  
      bottom[0]->num(),  
      bottom[0]->channels(),  
      Dtype(1.0),                                         
      diff_sq_ap_.gpu_data(),              
      summer_vec_.gpu_data(),                             
      Dtype(0.0),                                         
      dist_sq_ap_.mutable_gpu_data());  
  
  caffe_gpu_powx(  
        count,  
        diff_wn_.mutable_gpu_data(),  
        Dtype(2),  
        diff_sq_wn_.mutable_gpu_data());  
  caffe_gpu_gemv(  
        CblasNoTrans,  
        bottom[0]->num(),  
        bottom[0]->channels(),  
        Dtype(1.0),                                         
        diff_sq_wn_.gpu_data(),  
        summer_vec_.gpu_data(),                             
        Dtype(0.0),                                         
        dist_sq_wn_.mutable_gpu_data());  
  
  Dtype margin = this->layer_param_.triplet_loss_param().margin();  
  Dtype loss(0.0); 
  Dtype loss1(0.0);
  Dtype loss2(0.0); 
  Dtype Sam(0.0);
  Dtype unfaml(0.0);
  const Dtype* sampleW = bottom[4]->gpu_data();												//1111
  for (int i = 0; i < bottom[0]->num(); ++i) { 
	loss1 +=  std::max(Dtype(0.05) - margin +dist_sq_ap_.cpu_data()[i], Dtype(0.0));
	Sam +=dist_sq_ap_.cpu_data()[i];
  }
   for (int i = 0; i < bottom[0]->num(); ++i) { 
	unfaml +=dist_sq_wn_.cpu_data()[i];
	loss2 +=  std::max(Dtype(0.05) + margin -dist_sq_wn_.cpu_data()[i], Dtype(0.0));
  }
	loss = loss1 + loss2 ;
 
  loss = loss / static_cast<Dtype>(bottom[0]->num());
  top[0]->mutable_cpu_data()[0] = loss /Dtype(2) ;  
}  

template <typename Dtype>  
__global__ void CLLBackward(const int count, const int channels,  
    const Dtype margin, const Dtype alpha, const Dtype* sampleW,  
    const Dtype* diff, const Dtype* dist_sq_ap_, const Dtype* dist_sq_wn_,  
    Dtype *bottom_diff,const Dtype type) {  
	if (type == 1){									
		  CUDA_KERNEL_LOOP(i, count) {  
    		  int n = i / channels;  
   		  Dtype mdist(0.0);  
    		  mdist = Dtype(0.05) - margin + dist_sq_ap_[n] ;  
              	  if (mdist > 0.0) {  
        		  bottom_diff[i] =  alpha*diff[i];  
				
                   } else {  
        		   bottom_diff[i] = 0;  
     			}  
  		  } 
	} 
	if (type == 0){									
		  CUDA_KERNEL_LOOP(i, count) {  
    		  int n = i / channels;  
   		  Dtype mdist(0.0);  
    		  mdist = Dtype(0.05) + margin - dist_sq_wn_[n];  
              	  if (mdist > 0.0) {  
        		  bottom_diff[i] =  alpha*diff[i];  
                   } else {  
        		   bottom_diff[i] = 0;  
     			}  
  		  } 
	} 
	
}    
  
template <typename Dtype>  
void PairwiseLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,  
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {  
  Dtype margin = this->layer_param_.triplet_loss_param().margin();  
  const int count = bottom[0]->count();  
  const int channels = bottom[0]->channels();  
  
  for (int i = 0; i < 4; ++i) {  
    if (propagate_down[i]) { 
      const Dtype type = (i<2) ? 1 : 0; 
      const Dtype sign = ((i<=2)&&(i>=1)) ? -1 : 1;				
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /  	
          static_cast<Dtype>(bottom[0]->num());  
      if(i==0){  
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              bottom[4]->gpu_data(),  
              diff_ap_.gpu_data(),  
              dist_sq_ap_.gpu_data(),  
              dist_sq_wn_.gpu_data(),  
              bottom[i]->mutable_gpu_diff(),
		type);  
          CUDA_POST_KERNEL_CHECK;  
      }else if(i==1){  
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              bottom[4]->gpu_data(),  
              diff_ap_.gpu_data(),  
              dist_sq_ap_.gpu_data(),  
              dist_sq_wn_.gpu_data(),  
              bottom[i]->mutable_gpu_diff(),
		type);  
          CUDA_POST_KERNEL_CHECK;  
      }else if(i==2){  
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              bottom[4]->gpu_data(),  
              diff_wn_.gpu_data(),  
              dist_sq_ap_.gpu_data(),  
              dist_sq_wn_.gpu_data(), 
              bottom[i]->mutable_gpu_diff(),
		type);  
          CUDA_POST_KERNEL_CHECK;  
  
      } else if(i==3){  
          // NOLINT_NEXT_LINE(whitespace/operators)  
          CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(  
              count, channels, margin, alpha,  
              bottom[4]->gpu_data(),  
              diff_wn_.gpu_data(),  
              dist_sq_ap_.gpu_data(),  
              dist_sq_wn_.gpu_data(),  
              bottom[i]->mutable_gpu_diff(),
		type);  
          CUDA_POST_KERNEL_CHECK;  

  
      }  
    }  
  }  
}  
  
INSTANTIATE_LAYER_GPU_FUNCS(PairwiseLossLayer);  
  
}  // namespace caffe 
